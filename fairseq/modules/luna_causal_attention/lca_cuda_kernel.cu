#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include "THC/THC.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <math.h>
#include <vector>
#include <stdio.h>

#define FULL_MASK 0xffffffff


template<typename scalar_t>
__global__ 
void lca_cuda_forward_kernel(
    const scalar_t* __restrict__ x,
    const scalar_t* __restrict__ y,
    const scalar_t* __restrict__ z,
    scalar_t* __restrict__ f) {


}

torch::Tensor lca_cuda_forward(
    torch::Tensor const& x,
    torch::Tensor const& y,
    torch::Tensor const& z) {
    /*
     Args:
         x: [len, bsz, dim1]
         y: [len, bsz, dim1]
         z: [len, bsz, dim2]
     return:
         f: [len, bsz, dim2]
    */
    
    const int len = x.size(0);
    const int bsz = x.size(1);
    const int dim_xy = x.size(2);
    const int dim_z = z.size(2);

    const int xy_inc_t = bsz * dim_xy;
    const int z_inc_t = bsz * dim_z;

    auto act_options  = x.options().requires_grad(false);
    torch::Tensor f = torch::zeros({len, bsz, dim_z}, act_options);

    
}